#include <iostream>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"

static const unsigned int BLOCK_DIM = 16;
static const unsigned int RESET_DIM = BLOCK_DIM * BLOCK_DIM;

static int render_width, render_height, render_n;
static int render_count;
static float* render_buffer;
static float* dev_render_buffer;

static hiprandState* dev_curand_state;

__global__ void resetRenderKernel(hiprandState* curand_state, int render_res) {

	int t = blockDim.x * blockIdx.x + threadIdx.x;

	if (t < render_res) {
		hiprand_init(t, 0, 0, &curand_state[t]);
	}

}

__global__ void renderKernel(float* render_buffer, hiprandState* curand_state, int render_width, int render_height) {

	int x = BLOCK_DIM * blockIdx.x + threadIdx.x;
	int y = BLOCK_DIM * blockIdx.y + threadIdx.y;

	if (x < render_width && y < render_height) {

		int t = render_height * x + y;

		render_buffer[render_height * 3 * x + 3 * y + 0] += hiprand_uniform(&curand_state[t]);
		render_buffer[render_height * 3 * x + 3 * y + 1] += hiprand_uniform(&curand_state[t]);
		render_buffer[render_height * 3 * x + 3 * y + 2] += hiprand_uniform(&curand_state[t]);

	}

}

bool clearRenderBuffer() {

	if (hipMemset(dev_render_buffer, 0, render_n * sizeof(float)) != hipSuccess) {
		std::cout << "Cannot clear render buffer." << std::endl;
		return false;
	}

	return true;

}

bool downloadRenderBuffer() {

	if (hipMemcpy(render_buffer, dev_render_buffer, render_n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "Cannot download render buffer." << std::endl;
		return false;
	}

	return true;

}

bool resetRender(int width, int height) {

	render_width = width;
	render_height = height;
	render_n = render_width * render_height * 3;
	render_count = 0;

	if (render_buffer != nullptr) {
		delete render_buffer;
	}
	render_buffer = new float[render_n];

	if (hipSetDevice(0) != hipSuccess) {
		std::cout << "Cannot find CUDA device." << std::endl;
		return false;
	}

	if (hipMalloc(&dev_render_buffer, render_n * sizeof(float)) != hipSuccess) {
		std::cout << "Cannot allocate enough GPU memory." << std::endl;
		return false;
	}

	if (!clearRenderBuffer()) {
		hipFree(dev_render_buffer);
		return false;
	}

	if (hipMalloc(&dev_curand_state, render_width * render_height * sizeof(hiprandState)) != hipSuccess) {
		std::cout << "Cannot allocate enough GPU memory." << std::endl;
		hipFree(dev_render_buffer);
		return false;
	}

	int blocks = (render_width * render_height + RESET_DIM - 1) / RESET_DIM;
	int threads_per_block = RESET_DIM;
	resetRenderKernel<<<blocks, threads_per_block>>>(dev_curand_state, render_width * render_height);
	hipError_t cudaStatus;

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error launching render kernel: " << hipGetErrorString(cudaStatus) << std::endl;
		hipFree(dev_render_buffer);
		hipFree(dev_curand_state);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error synchronizing with device: " << hipGetErrorString(cudaStatus) << std::endl;
		hipFree(dev_render_buffer);
		hipFree(dev_curand_state);
		return false;
	}

	return true;

}

bool render(unsigned char* image_data) {

	render_count++;

	dim3 blocks((render_width + BLOCK_DIM - 1) / BLOCK_DIM, (render_height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
	renderKernel<<<blocks, threads_per_block>>>(dev_render_buffer, dev_curand_state, render_width, render_height);

	hipError_t cudaStatus;

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error launching render kernel: " << hipGetErrorString(cudaStatus) << std::endl;
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error synchronizing with device: " << hipGetErrorString(cudaStatus) << std::endl;
		return false;
	}

	if (!downloadRenderBuffer()) {
		return false;
	}

	for (int i = 0; i < render_width * render_height * 3; i++) {
		image_data[i] = fminf((256.0f * render_buffer[i]) / render_count, 255.0f);
	}

	return true;

}
