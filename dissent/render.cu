#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include <iostream>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include "common.h"
#include "geometry.h"
#include "scene.h"

static const unsigned int BLOCK_DIM = 16;
static const unsigned int RESET_DIM = BLOCK_DIM * BLOCK_DIM;

static int render_width, render_height, render_n;
static int render_count;
static bool should_clear = false;

static color3* render_buffer;
static color3* dev_render_buffer;

static sphere_t* dev_spheres;
static surface_t* dev_surfaces;

static hiprandState* dev_curand_state;

__device__ int kernel_render_width, kernel_render_height, kernel_render_n;
__device__ color3* kernel_render_buffer;
__device__ scene_parameters_t kernel_scene_params;
__device__ int kernel_num_spheres;
__device__ sphere_t* kernel_spheres;
__device__ surface_t* kernel_surfaces;
__device__ hiprandState* kernel_curand_state;

__device__ bool sphere_t::intersect(vec3 start, vec3 direction, float& t, vec3& normal, bool& exiting) {

	float a = direction.magnitude_2();
	vec3 recentered = start - center;
	float b = 2 * direction.dot(recentered);
	float recentered_radius_2 = recentered.magnitude_2();
	float c = recentered_radius_2 - (radius * radius);

	float discrim = (b * b) - (4.0f * a * c);
	if (discrim < 0.0f) return false;

	float sqrt_discrim = std::sqrtf(discrim);
	float t1 = (-b + sqrt_discrim) / (2.0f * a);
	float t2 = (-b - sqrt_discrim) / (2.0f * a);

	exiting = recentered_radius_2 < radius * radius;

	if (exiting) {
		t = std::fmaxf(t1, t2);
	} else {
		t = std::fminf(t1, t2);
	}
	if (t < 0.0f) return false;

	vec3 surface_point = start + direction * t;
	normal = (surface_point - center) / radius;
	if (exiting) normal = -normal;

	return true;

}

__device__ vec3 random_hemi_normal(vec3 normal, int n) {
	vec3 hemi;
	do {
		hemi.x = 2.0f * hiprand_uniform(&kernel_curand_state[n]) - 1.0f;
		hemi.y = 2.0f * hiprand_uniform(&kernel_curand_state[n]) - 1.0f;
		hemi.z = 2.0f * hiprand_uniform(&kernel_curand_state[n]) - 1.0f;
	} while (hemi.magnitude_2() > 1);
	hemi.normalize();
	if (hemi.dot(normal) < 0.0f) {
		hemi = -hemi;
	}
	return hemi;
}

__device__ vec3 confusion_disk(vec3 ortho1, vec3 ortho2, int n) {
	float theta = 2.0f * M_PI * hiprand_uniform(&kernel_curand_state[n]);
	float sqrtr = sqrtf(hiprand_uniform(&kernel_curand_state[n]));
	return ortho1 * sqrtr * sinf(theta) + ortho2 * sqrtr * cosf(theta);
}

__global__ void resetRenderKernel(color3* render_buffer, hiprandState* curand_state, sphere_t* spheres, surface_t* surfaces, int render_width, int render_height, scene_parameters_t scene_params, int num_spheres) {

	int t = blockDim.x * blockIdx.x + threadIdx.x;
	int render_n = render_width * render_height;

	if (t == 0) {
		kernel_render_width = render_width;
		kernel_render_height = render_height;
		kernel_render_n = render_n;
		kernel_render_buffer = render_buffer;
		kernel_scene_params = scene_params;
		kernel_num_spheres = num_spheres;
		kernel_spheres = spheres;
		kernel_surfaces = surfaces;
		kernel_curand_state = curand_state;
	}

	if (t < render_n) {
		hiprand_init(t, 0, 0, &curand_state[t]);
	}

}

__global__ void renderKernel(camera_t camera) {

	int x = BLOCK_DIM * blockIdx.x + threadIdx.x;
	int y = BLOCK_DIM * blockIdx.y + threadIdx.y;

	if (x < kernel_render_width && y < kernel_render_height) {

		int n = kernel_render_height * x + y;

		float screen_x = (x + hiprand_uniform(&kernel_curand_state[n]) - 0.5f) / kernel_render_width - 0.5f;
		float screen_y = (y + hiprand_uniform(&kernel_curand_state[n]) - 0.5f) / kernel_render_height - 0.5f;
		vec3 dof_confusion = confusion_disk(camera.up, camera.right, n) * kernel_scene_params.aperture_radius;
		vec3 ray_start = camera.position + dof_confusion;
		vec3 ray_direction = (camera.forward + camera.right * camera.aspect_ratio * screen_x + camera.up * screen_y) * kernel_scene_params.focal_distance - dof_confusion;
		ray_direction.normalize();

		color3 final_color = {0.0f, 0.0f, 0.0f};
		color3 d_product = {1.0f, 1.0f, 1.0f};

		for (int depth = 0; depth < 15; depth++) {

			float t;
			vec3 normal;
			bool exiting;

			float best_t = INFINITY;
			vec3 best_normal;
			int best_surface;
			bool best_exiting;

			for (int i = 0; i < kernel_num_spheres; i++) {
				if (kernel_spheres[i].intersect(ray_start, ray_direction, t, normal, exiting)) {
					if (t < best_t) {
						best_t = t;
						best_normal = normal;
						best_surface = i;
						best_exiting = exiting;
					}
				}
			}

			if (best_t < INFINITY) {

				ray_start = ray_start + ray_direction * best_t;
				vec3 off_surface = best_normal * 0.0001f;

				float effective_reflectance;
				float effective_transmission;

				float n1 = best_exiting ? kernel_surfaces[best_surface].refractive_index : 1.0f;
				float n2 = best_exiting ? 1.0f : kernel_surfaces[best_surface].refractive_index;
				float ni = n1 / n2;

				float cosi = -ray_direction.dot(best_normal);
				float sint_2 = ni * ni * (1 - cosi * cosi);
				float cost = sqrtf(1 - sint_2);

				if (kernel_surfaces[best_surface].reflectance > 0.0f) {

					if (sint_2 > 1) {
						effective_reflectance = 1.0f;
					} else {
						float r0 = (n1 - n2) / (n1 + n2);
						r0 *= r0;
						float base;
						if (n1 <= n2) {
							base = 1.0f - cosi;
						} else {
							base = 1.0f - cost;
						}
						float r_schlick = r0 + (1 - r0) * base * base * base * base * base;
						effective_reflectance = kernel_surfaces[best_surface].reflectance + (1.0f - kernel_surfaces[best_surface].reflectance) * r_schlick;
					}
				} else {
					effective_reflectance = 0.0f;
				}

				if (hiprand_uniform(&kernel_curand_state[n]) < effective_reflectance) {

					ray_start += off_surface;
					ray_direction = ray_direction.reflect(best_normal);

					final_color += d_product * kernel_surfaces[best_surface].emit;
					d_product *= kernel_surfaces[best_surface].specular;

				} else if (hiprand_uniform(&kernel_curand_state[n]) < kernel_surfaces[best_surface].transmission) {

					ray_start -= off_surface;
					ray_direction = ray_direction * ni + best_normal * (ni * cosi - cost);
					ray_direction.normalize();
					if (best_exiting) {
						color3 attenuation_color = kernel_surfaces[best_surface].attenuation_color;
						color3 beer = {expf(best_t * logf(attenuation_color.r)), expf(best_t * logf(attenuation_color.g)), expf(best_t * logf(attenuation_color.b))};
						d_product *= beer;
					}

				} else {

					ray_start += off_surface;
					ray_direction = random_hemi_normal(best_normal, n);

					final_color += d_product * kernel_surfaces[best_surface].emit;
					d_product *= kernel_surfaces[best_surface].diffuse;

				}
			} else {

				final_color += d_product * kernel_scene_params.background_emission;
				break;
			}

		}

		kernel_render_buffer[kernel_render_width * y + x] += final_color;

	}

}

bool clearRenderBuffer() {

	if (hipMemset(dev_render_buffer, 0, render_n * sizeof(color3)) != hipSuccess) {
		std::cout << "Cannot clear render buffer." << std::endl;
		return false;
	}

	return true;

}

bool downloadRenderBuffer() {

	if (hipMemcpy(render_buffer, dev_render_buffer, render_n * sizeof(color3), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "Cannot download render buffer." << std::endl;
		return false;
	}

	return true;

}

bool resetRender(int width, int height, scene_t& scene) {

	render_width = width;
	render_height = height;
	render_n = render_width * render_height;
	render_count = 0;

	if (render_buffer != nullptr) {
		delete render_buffer;
	}
	render_buffer = new color3[render_n];

	if (hipSetDevice(0) != hipSuccess) {
		std::cout << "Cannot find CUDA device." << std::endl;
		return false;
	}

	if (hipMalloc(&dev_render_buffer, render_n * sizeof(color3)) != hipSuccess) {
		std::cout << "Cannot allocate enough GPU memory." << std::endl;
		return false;
	}

	if (!clearRenderBuffer()) {
		return false;
	}

	if (hipMalloc(&dev_spheres, scene.spheres.size() * sizeof(sphere_t)) != hipSuccess) {
		std::cout << "Cannot allocate enough GPU memory." << std::endl;
		return false;
	}

	if (hipMemcpy(dev_spheres, scene.spheres.data(), scene.spheres.size() * sizeof(sphere_t), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "Cannot upload spheres." << std::endl;
		return false;
	}

	if (hipMalloc(&dev_surfaces, scene.spheres.size() * sizeof(surface_t)) != hipSuccess) {
		std::cout << "Cannot allocate enough GPU memory." << std::endl;
		return false;
	}

	if (hipMemcpy(dev_surfaces, scene.surfaces.data(), scene.spheres.size() * sizeof(surface_t), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "Cannot upload surfaces." << std::endl;
		return false;
	}

	if (hipMalloc(&dev_curand_state, render_n * sizeof(hiprandState)) != hipSuccess) {
		std::cout << "Cannot allocate enough GPU memory." << std::endl;
		return false;
	}

	int blocks = (render_n + RESET_DIM - 1) / RESET_DIM;
	int threads_per_block = RESET_DIM;
	resetRenderKernel<<<blocks, threads_per_block>>>(dev_render_buffer, dev_curand_state, dev_spheres, dev_surfaces, render_width, render_height, scene.params, scene.spheres.size());
	hipError_t cudaStatus;

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error launching render kernel: " << hipGetErrorString(cudaStatus) << std::endl;
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error synchronizing with device: " << hipGetErrorString(cudaStatus) << std::endl;
		return false;
	}

	return true;

}

bool render(unsigned char* image_data, camera_t& camera) {

	if (should_clear) {
		clearRenderBuffer();
		render_count = 0;
		should_clear = false;
	}

	render_count++;

	dim3 blocks((render_width + BLOCK_DIM - 1) / BLOCK_DIM, (render_height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
	renderKernel<<<blocks, threads_per_block>>>(camera);

	hipError_t cudaStatus;

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error launching render kernel: " << hipGetErrorString(cudaStatus) << std::endl;
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error synchronizing with device: " << hipGetErrorString(cudaStatus) << std::endl;
		return false;
	}

	if (!downloadRenderBuffer()) {
		return false;
	}

	for (int i = 0; i < render_n; i++) {
		color3 color = render_buffer[i] * (256.0f / render_count);
		image_data[3 * i] = fminf(color.r, 255.0f);
		image_data[3 * i + 1] = fminf(color.g, 255.0f);
		image_data[3 * i + 2] = fminf(color.b, 255.0f);
	}

	std::cout << render_count << std::endl;

	return true;

}

void clearRender() {

	render_count = 0;
	should_clear = true;

}
