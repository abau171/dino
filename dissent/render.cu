#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include "common.h"
#include "geometry.h"
#include "scene.h"

static const unsigned int BLOCK_DIM = 16;
static const unsigned int RESET_DIM = BLOCK_DIM * BLOCK_DIM;

static int render_width, render_height, render_n;
static int render_count;
static float* render_buffer;
static float* dev_render_buffer;

static hiprandState* dev_curand_state;

__device__ int kernel_render_width, kernel_render_height;
__device__ float* kernel_render_buffer;
__device__ hiprandState* kernel_curand_state;

__device__ bool sphere_t::intersect(vec3 start, vec3 direction, float& t, vec3& normal) {

	float a = direction.magnitude_2();
	vec3 recentered = start - center;
	float b = 2 * direction.dot(recentered);
	float c = recentered.magnitude_2() - (radius * radius);

	float discrim = (b * b) - (4.0f * a * c);
	if (discrim < 0.0f) return false;

	float sqrt_discrim = std::sqrtf(discrim);
	float t1 = (-b + sqrt_discrim) / (2.0f * a);
	float t2 = (-b - sqrt_discrim) / (2.0f * a);

	if (t1 > 0.0f && t2 > 0.0f) {
		t = std::fminf(t1, t2);
	} else {
		t = std::fmaxf(t1, t2);
	}
	if (t < 0.0f) return false;

	vec3 surface_point = start + direction * t;
	normal = (surface_point - center) / radius;

	return true;

}

__global__ void resetRenderKernel(float* render_buffer, hiprandState* curand_state, int render_width, int render_height) {

	int t = blockDim.x * blockIdx.x + threadIdx.x;

	if (t == 0) {
		kernel_render_width = render_width;
		kernel_render_height = render_height;
		kernel_render_buffer = render_buffer;
		kernel_curand_state = curand_state;
	}

	if (t < render_width * render_height) {
		hiprand_init(t, 0, 0, &curand_state[t]);
	}

}

__global__ void renderKernel() {

	int x = BLOCK_DIM * blockIdx.x + threadIdx.x;
	int y = BLOCK_DIM * blockIdx.y + threadIdx.y;

	if (x < kernel_render_width && y < kernel_render_height) {

		int n = kernel_render_height * x + y;

		vec3 position = {0.0f, 5.0f, 7.0f};

		vec3 lookat = {0.0f, 2.11f, 0.0f};
		vec3 forward = (lookat - position);
		forward.normalize();
		vec3 up = {0.0f, 1.0f, 0.0f};
		vec3 right = forward.cross(up);
		right.normalize();
		up = right.cross(forward);

		camera_t camera = {
			position,
			forward,
			up,
			right,
			(float) kernel_render_width / kernel_render_height
		};

		float screen_x = (float) x / kernel_render_width - 0.5f;
		float screen_y = (float) y / kernel_render_height - 0.5f;
		vec3 ray_direction = camera.forward + (camera.right * camera.aspect_ratio * screen_x + camera.up * screen_y);
		ray_direction.normalize();

		sphere_t sphere = {{1.0f, 2.5f, 0.5f}, 1.5f};

		float t;
		vec3 normal;
		float out = 0.0f;
		if (sphere.intersect(camera.position, ray_direction, t, normal)) {
			out = 1.0f;
		}

		kernel_render_buffer[kernel_render_width * 3 * y + 3 * x + 0] += out;
		kernel_render_buffer[kernel_render_width * 3 * y + 3 * x + 1] += out;
		kernel_render_buffer[kernel_render_width * 3 * y + 3 * x + 2] += out;

	}

}

bool clearRenderBuffer() {

	if (hipMemset(dev_render_buffer, 0, render_n * sizeof(float)) != hipSuccess) {
		std::cout << "Cannot clear render buffer." << std::endl;
		return false;
	}

	return true;

}

bool downloadRenderBuffer() {

	if (hipMemcpy(render_buffer, dev_render_buffer, render_n * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "Cannot download render buffer." << std::endl;
		return false;
	}

	return true;

}

bool resetRender(int width, int height) {

	render_width = width;
	render_height = height;
	render_n = render_width * render_height * 3;
	render_count = 0;

	if (render_buffer != nullptr) {
		delete render_buffer;
	}
	render_buffer = new float[render_n];

	if (hipSetDevice(0) != hipSuccess) {
		std::cout << "Cannot find CUDA device." << std::endl;
		return false;
	}

	if (hipMalloc(&dev_render_buffer, render_n * sizeof(float)) != hipSuccess) {
		std::cout << "Cannot allocate enough GPU memory." << std::endl;
		return false;
	}

	if (!clearRenderBuffer()) {
		hipFree(dev_render_buffer);
		return false;
	}

	if (hipMalloc(&dev_curand_state, render_width * render_height * sizeof(hiprandState)) != hipSuccess) {
		std::cout << "Cannot allocate enough GPU memory." << std::endl;
		hipFree(dev_render_buffer);
		return false;
	}

	int blocks = (render_width * render_height + RESET_DIM - 1) / RESET_DIM;
	int threads_per_block = RESET_DIM;
	resetRenderKernel<<<blocks, threads_per_block>>>(dev_render_buffer, dev_curand_state, render_width, render_height);
	hipError_t cudaStatus;

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error launching render kernel: " << hipGetErrorString(cudaStatus) << std::endl;
		hipFree(dev_render_buffer);
		hipFree(dev_curand_state);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error synchronizing with device: " << hipGetErrorString(cudaStatus) << std::endl;
		hipFree(dev_render_buffer);
		hipFree(dev_curand_state);
		return false;
	}

	return true;

}

bool render(unsigned char* image_data) {

	render_count++;

	dim3 blocks((render_width + BLOCK_DIM - 1) / BLOCK_DIM, (render_height + BLOCK_DIM - 1) / BLOCK_DIM);
	dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
	renderKernel<<<blocks, threads_per_block>>>();

	hipError_t cudaStatus;

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error launching render kernel: " << hipGetErrorString(cudaStatus) << std::endl;
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		std::cout << "Error synchronizing with device: " << hipGetErrorString(cudaStatus) << std::endl;
		return false;
	}

	if (!downloadRenderBuffer()) {
		return false;
	}

	for (int i = 0; i < render_width * render_height * 3; i++) {
		image_data[i] = fminf((256.0f * render_buffer[i]) / render_count, 255.0f);
	}

	return true;

}
